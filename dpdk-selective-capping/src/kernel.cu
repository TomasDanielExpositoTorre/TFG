#include "hip/hip_runtime.h"
#include "headers.h"

__global__ void selective_capping(struct rte_gpu_comm_list *global_list,
                                  struct pcap_packet_header *headers,
                                  struct arguments args)
{
    int id = threadIdx.x;
    int block = blockIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    struct rte_gpu_comm_list *comm_list = &(global_list[block]);
    int runlen = 0, total = 0;
    int psize;
    char *packet;

    if (id < comm_list->num_pkts)
    {
        packet = (char *)(comm_list->pkt_list[id].addr);
        psize = comm_list->pkt_list[id].size;

        /* Broadcast capture time through burst*/
        headers[i].ts_sec = headers[i].ts_sec;
        headers[i].ts_usec = headers[i].ts_usec;
        headers[i].len = psize;

        for (int j = MIN_HLEN; j < psize; j++)
        {
            if (packet[j] >= MIN_ASCII && packet[j] <= MAX_ASCII)
            {
                runlen++;
                total += 100;
                if (runlen == args.ascii_runlen)
                    j = psize;
            }
            else
                runlen = 0;
        }

        if (MAX_HLEN > psize || runlen == args.ascii_runlen || total >= (args.ascii_percentage * (psize - MIN_HLEN)))
            headers[i].caplen = psize;
        else
            headers[i].caplen = MAX_HLEN;
    }

    __syncthreads();

    if (id == 0)
        *(comm_list->status_d) = RTE_GPU_COMM_LIST_DONE;
}

__global__ void optimized_capping(struct rte_gpu_comm_list *global_list,
                                  struct pcap_packet_header *headers,
                                  struct arguments args)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    struct rte_gpu_comm_list *comm_list = &(global_list[blockIdx.x]);
    int runlen = 0, total = 0, seen = 0;
    int psize;
    char *packet;

    if (threadIdx.x < comm_list->num_pkts)
    {
        packet = (char *)(comm_list->pkt_list[threadIdx.x].addr);
        psize = comm_list->pkt_list[threadIdx.x].size;

        /* Broadcast capture time through burst*/
        headers[i].ts_sec = headers[i].ts_sec;
        headers[i].ts_usec = headers[i].ts_usec;
        headers[i].len = psize;

        for (int j = MIN_HLEN + args.ascii_runlen - 1; j >= MIN_HLEN && j < psize; j--, seen++)
        {
            if (packet[j] >= MIN_ASCII && packet[j] <= MAX_ASCII)
            {
                runlen++;
                total += 100;
                if (runlen == args.ascii_runlen)
                    j = psize;
            }
            else
            {
                runlen = 0;
                j += args.ascii_runlen + 1;
            }
        }

        if (MAX_HLEN > psize || runlen == args.ascii_runlen || total >= (args.ascii_percentage * seen))
            headers[i].caplen = psize;
        else
            headers[i].caplen = MAX_HLEN;
    }

    __syncthreads();

    if (threadIdx.x == 0)
        *(comm_list->status_d) = RTE_GPU_COMM_LIST_DONE;
}

void launch_kernel(struct rte_gpu_comm_list *comm_list, int blocks, int threads,
                   hipStream_t stream, struct arguments args,
                   struct pcap_packet_header *headers)
{
    if (args.kernel == SELECTIVE_CAPPING)
        selective_capping<<<blocks, threads, 0, stream>>>(comm_list, headers, args);
    else
        optimized_capping<<<blocks, threads, 0, stream>>>(comm_list, headers, args);
}