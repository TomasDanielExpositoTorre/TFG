#include "hip/hip_runtime.h"
#include "headers.h"

__global__ void vanilla_capping_thread(struct rte_gpu_comm_list *comm_list,
                                       struct pcap_packet_header *headers,
                                       struct arguments args)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int psize;
    int runlen = 0, total = 0;
    char *packet;

    if (i < comm_list->num_pkts)
    {
        packet = (char *)(comm_list->pkt_list[i].addr);
        psize = comm_list->pkt_list[i].size;

        /* Broadcast capture time through burst*/
        headers[i] = headers[0];
        headers[i].len = psize;
        for (int j = MIN_HLEN; j < psize; j++)
        {
            if (packet[j] >= MIN_ASCII && packet[j] <= MAX_ASCII)
            {
                runlen++;
                total += 100;
                if (runlen == args.ascii_runlen)
                    j = psize;
            }
            else
                runlen = 0;
        }

        if (MAX_HLEN > psize || runlen == args.ascii_runlen || total >= (args.ascii_percentage * (psize - MIN_HLEN)))
            headers[i].caplen = psize;
        else
            headers[i].caplen = MAX_HLEN;
    }

    __syncthreads();

    if (i == 0)
        *(comm_list->status_d) = RTE_GPU_COMM_LIST_DONE;
}

__global__ void optimized_capping_thread(struct rte_gpu_comm_list *comm_list,
                                         struct pcap_packet_header *headers,
                                         struct arguments args)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int runlen = 0, total = 0, seen = 0;
    int packetlen;
    char *packet;

    if (i < comm_list->num_pkts)
    {
        packet = (char *)(comm_list->pkt_list[i].addr);
        packetlen = comm_list->pkt_list[i].size;
        comm_list->pkt_list[i].size <<= 1;

        /* Don't cap if payload is less than runlen */
        if (MIN_HLEN + args.ascii_runlen > comm_list->pkt_list[i].size)
            return;

        for (int j = MIN_HLEN + args.ascii_runlen - 1; j >= MIN_HLEN && j < packetlen; j--, seen++)
        {
            if (packet[j] >= MIN_ASCII && packet[j] <= MAX_ASCII)
            {
                runlen++;
                total += 100;
                if (runlen == args.ascii_runlen)
                    return; /* Do not cap */
            }
            else
            {
                runlen = 0;
                j += args.ascii_runlen + 1;
            }
        }

        comm_list->pkt_list[i].size |= (total < (args.ascii_percentage * seen)); /* Cap to MAX_HLEN bytes */
    }

    __syncthreads();

    if (i == 0)
        *(comm_list->status_d) = RTE_GPU_COMM_LIST_DONE;
}

__global__ void naive_capping_warp(struct rte_gpu_comm_list *comm_list, struct arguments args)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    printf("%d\n", i);
}

__global__ void invasive_capping_warp(struct rte_gpu_comm_list *comm_list, struct arguments args)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    printf("%d\n", i);
}

__global__ void coercive_capping_warp(struct rte_gpu_comm_list *comm_list, struct arguments args)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    printf("%d\n", i);
}

void launch_kernel(struct rte_gpu_comm_list *comm_list, int blocks, int threads,
                   hipStream_t stream, struct arguments args,
                   struct pcap_packet_header *headers)
{
    if (args.kernel == VANILLA_CAPPING_THREAD)
        vanilla_capping_thread<<<blocks, threads, 0, stream>>>(comm_list, headers, args);
    else if (args.kernel == OPTIMIZED_CAPPING_THREAD)
        optimized_capping_thread<<<blocks, threads, 0, stream>>>(comm_list, headers, args);
    else if (args.kernel == NAIVE_CAPPING_WARP)
        naive_capping_warp<<<blocks, threads, 0, stream>>>(comm_list, args);
    else if (args.kernel == INVASIVE_CAPPING_WARP)
        invasive_capping_warp<<<blocks, threads, 0, stream>>>(comm_list, args);
    else
        coercive_capping_warp<<<blocks, threads, 0, stream>>>(comm_list, args);
}